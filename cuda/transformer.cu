
#include <hip/hip_runtime.h>
// Error checking macro
#define CUDA_CHECK(call) \
  do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(error)); \
      exit(EXIT_FAILURE); \
    } \
  } while(0)

// Memory allocation helper
template <typename T>
T* allocateDeviceMemory(size_t size) {
    T* ptr;
    CUDA_CHECK(hipMalloc(&ptr, size * sizeof(T)));
    return ptr;
}

// Memory copy helper
template <typename T>
void copyToDevice(T* dst, const T* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size * sizeof(T), hipMemcpyHostToDevice));
}